/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2019, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/lower_trs_kernels.hpp"


#include <functional>
#include <memory>


#include <hip/hip_runtime.h>
#include <hipsparse.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/solver/lower_trs.hpp>


#include "core/matrix/dense_kernels.hpp"
#include "core/solver/lower_trs_kernels.hpp"
#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/device_guard.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The LOWER_TRS solver namespace.
 *
 * @ingroup lower_trs
 */
namespace lower_trs {


void perform_transpose(std::shared_ptr<const CudaExecutor> exec,
                       bool &transposability)
{
#if (defined(CUDA_VERSION) && (CUDA_VERSION >= 9020))
    transposability = false;
#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9020))
    transposability = true;
#endif
}


void init_struct(std::shared_ptr<const CudaExecutor> exec,
                 std::shared_ptr<gko::solver::SolveStruct> &solve_struct)
{
    const auto id = exec->get_device_id();
    device_guard g(id);
    solve_struct = std::shared_ptr<gko::solver::SolveStruct>(
        kernels::cuda::cusparse::init_trs_solve_struct(),
        [id](gko::solver::SolveStruct *solve_struct_) {
            device_guard g(id);
            kernels::cuda::cusparse::clear_trs_solve_struct(solve_struct_);
        });
}


template <typename ValueType, typename IndexType>
void generate(std::shared_ptr<const CudaExecutor> exec,
              const matrix::Csr<ValueType, IndexType> *matrix,
              gko::solver::SolveStruct *solve_struct,
              const gko::size_type num_rhs)
{
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        auto handle = exec->get_cusparse_handle();
#if (defined(CUDA_VERSION) && (CUDA_VERSION >= 9020))
        ValueType one = 1.0;

        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::buffer_size_ext(
            handle, solve_struct->algorithm, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_TRANSPOSE, matrix->get_size()[0], num_rhs,
            matrix->get_num_stored_elements(), &one, solve_struct->factor_descr,
            matrix->get_const_values(), matrix->get_const_row_ptrs(),
            matrix->get_const_col_idxs(), nullptr, num_rhs,
            solve_struct->solve_info, solve_struct->policy,
            &solve_struct->factor_work_size);

        // allocate workspace
        solve_struct->factor_work_vec =
            exec->alloc<void *>(solve_struct->factor_work_size);

        cusparse::csrsm2_analysis(
            handle, solve_struct->algorithm, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_TRANSPOSE, matrix->get_size()[0], num_rhs,
            matrix->get_num_stored_elements(), &one, solve_struct->factor_descr,
            matrix->get_const_values(), matrix->get_const_row_ptrs(),
            matrix->get_const_col_idxs(), nullptr, num_rhs,
            solve_struct->solve_info, solve_struct->policy,
            solve_struct->factor_work_vec);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));

#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9020))

        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::csrsm_analysis(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->get_size()[0],
            matrix->get_num_stored_elements(), solve_struct->factor_descr,
            matrix->get_const_values(), matrix->get_const_row_ptrs(),
            matrix->get_const_col_idxs(), solve_struct->solve_info);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));
#endif
    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_LOWER_TRS_GENERATE_KERNEL);


template <typename ValueType, typename IndexType>
void solve(std::shared_ptr<const CudaExecutor> exec,
           const matrix::Csr<ValueType, IndexType> *matrix,
           gko::solver::SolveStruct *solve_struct,
           matrix::Dense<ValueType> *trans_b, matrix::Dense<ValueType> *trans_x,
           const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *x)
{
    using vec = matrix::Dense<ValueType>;
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        ValueType one = 1.0;
        auto handle = exec->get_cusparse_handle();
#if (defined(CUDA_VERSION) && (CUDA_VERSION >= 9020))
        x->copy_from(gko::lend(b));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::csrsm2_solve(
            handle, solve_struct->algorithm, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_TRANSPOSE, matrix->get_size()[0],
            b->get_stride(), matrix->get_num_stored_elements(), &one,
            solve_struct->factor_descr, matrix->get_const_values(),
            matrix->get_const_row_ptrs(), matrix->get_const_col_idxs(),
            x->get_values(), b->get_stride(), solve_struct->solve_info,
            solve_struct->policy, solve_struct->factor_work_vec);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));
#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9020))
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        if (b->get_stride() == 1) {
            auto temp_b = const_cast<ValueType *>(b->get_const_values());
            cusparse::csrsm_solve(
                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->get_size()[0],
                b->get_stride(), &one, solve_struct->factor_descr,
                matrix->get_const_values(), matrix->get_const_row_ptrs(),
                matrix->get_const_col_idxs(), solve_struct->solve_info, temp_b,
                b->get_size()[0], x->get_values(), x->get_size()[0]);
        } else {
            dense::transpose(exec, trans_b, b);
            dense::transpose(exec, trans_x, x);
            cusparse::csrsm_solve(
                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->get_size()[0],
                trans_b->get_size()[0], &one, solve_struct->factor_descr,
                matrix->get_const_values(), matrix->get_const_row_ptrs(),
                matrix->get_const_col_idxs(), solve_struct->solve_info,
                trans_b->get_values(), trans_b->get_size()[1],
                trans_x->get_values(), trans_x->get_size()[1]);
            dense::transpose(exec, x, trans_x);
        }
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));

#endif

    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_LOWER_TRS_SOLVE_KERNEL);


}  // namespace lower_trs
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
