/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2019, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/upper_trs_kernels.hpp"


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>


#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The UPPER_TRS solver namespace.
 *
 * @ingroup upper_trs
 */
namespace upper_trs {


template <typename ValueType, typename IndexType>
void generate(std::shared_ptr<const CudaExecutor> exec,
              const matrix::Csr<ValueType, IndexType> *matrix,
              const gko::size_type num_rhs) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_UPPER_TRS_GENERATE_KERNEL);


template <typename ValueType, typename IndexType>
void solve(std::shared_ptr<const CudaExecutor> exec,
           const matrix::Csr<ValueType, IndexType> *matrix,
           const matrix::Dense<ValueType> *b,
           matrix::Dense<ValueType> *x) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_UPPER_TRS_SOLVE_KERNEL);


}  // namespace upper_trs
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
